#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

            thrust::host_vector<int> h_in(idata, idata + n);
            thrust::device_vector<int> dv_in = h_in;
            thrust::device_vector<int> dv_out(n);
            thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            thrust::host_vector<int> h_out = dv_out;
            for (int i = 0; i < n; i++) {
                odata[i] = h_out[i];
            }
            timer().endGpuTimer();
        }
    }
}
