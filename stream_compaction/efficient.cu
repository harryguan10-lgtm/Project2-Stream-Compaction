#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void upSweep(int n, int* data, int stride) {
            int index = (blockIdx.x * blockDim.x + threadIdx.x) * (stride << 1);

            if (index + stride < n) {
                data[index + (stride << 1) - 1] += data[index + stride - 1];
            }
        }

      
        __global__ void downSweep(int n, int* data, int stride) {
            int index = (blockIdx.x * blockDim.x + threadIdx.x) * (stride << 1);

            if (index + stride < n) {
                int temp = data[index + stride - 1];
                data[index + stride - 1] = data[index + (stride << 1) - 1];
                data[index + (stride << 1) - 1] += temp;
            }
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int pow2_n = 1 << ilog2ceil(n);

            int* dev_data;
            hipMalloc((void**)&dev_data, pow2_n * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");

            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to device failed!");

            if (pow2_n > n) {
                hipMemset(dev_data + n, 0, (pow2_n - n) * sizeof(int));
                checkCUDAError("hipMemset failed!");
            }

            const int blockSize = 128;

            for (int stride = 1; stride < pow2_n; stride <<= 1) {
                int numThreads = pow2_n >> (ilog2(stride) + 1);
                if (numThreads > 0) {
                    dim3 blockDim(blockSize);
                    dim3 gridDim((numThreads + blockSize - 1) / blockSize);

                    upSweep << <gridDim, blockDim >> > (pow2_n, dev_data, stride);
                    checkCUDAError("upSweep failed!");
                }
            }

            hipMemset(dev_data + pow2_n - 1, 0, sizeof(int));
            checkCUDAError("hipMemset last element failed!");

            for (int stride = pow2_n >> 1; stride > 0; stride >>= 1) {
                int numThreads = pow2_n >> (ilog2(stride) + 1);
                if (numThreads > 0) {
                    dim3 blockDim(blockSize);
                    dim3 gridDim((numThreads + blockSize - 1) / blockSize);

                    downSweep << <gridDim, blockDim >> > (pow2_n, dev_data, stride);
                    checkCUDAError("downS failed!");
                }
            }

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from device failed!");

            hipFree(dev_data);

            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int* dev_idata, * dev_bools, * dev_indices, * dev_odata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_bools, n * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_indices, n * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to device failed!");

            const int blockSize = 128;
            dim3 blockDim(blockSize);
            dim3 gridDim((n + blockSize - 1) / blockSize);

            StreamCompaction::Common::kernMapToBoolean << <gridDim, blockDim >> > (n, dev_bools, dev_idata);
            checkCUDAError("kernMapToBoolean failed!");

            int* host_bools = new int[n];
            int* host_indices = new int[n];
            hipMemcpy(host_bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy bools to host failed!");

            scan(n, host_indices, host_bools);

            hipMemcpy(dev_indices, host_indices, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy indices to device failed!");

            StreamCompaction::Common::kernScatter << <gridDim, blockDim >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);
            checkCUDAError("kernScatter failed!");

            int count = host_indices[n - 1] + host_bools[n - 1];

            hipMemcpy(odata, dev_odata, count * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy result to host failed!");

            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_odata);

            timer().endGpuTimer();
            return count;
        }
    }
}
