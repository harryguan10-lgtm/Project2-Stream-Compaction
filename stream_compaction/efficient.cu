#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void upSweep(int n, int* data, int stride) {
            int index = (blockIdx.x * blockDim.x + threadIdx.x) * (stride << 1);

            if (index + stride < n) {
                data[index + (stride << 1) - 1] += data[index + stride - 1];
            }
        }

      
        __global__ void downSweep(int n, int* data, int stride) {
            int index = (blockIdx.x * blockDim.x + threadIdx.x) * (stride << 1);

            if (index + stride < n) {
                int temp = data[index + stride - 1];
                data[index + stride - 1] = data[index + (stride << 1) - 1];
                data[index + (stride << 1) - 1] += temp;
            }
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int pow2_n = 1 << ilog2ceil(n);

            int* dev_data;
            hipMalloc((void**)&dev_data, pow2_n * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");

            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to device failed!");

            if (pow2_n > n) {
                hipMemset(dev_data + n, 0, (pow2_n - n) * sizeof(int));
                checkCUDAError("hipMemset failed!");
            }

            const int blockSize = 128;

            for (int stride = 1; stride < pow2_n; stride <<= 1) {
                int numThreads = pow2_n >> (ilog2(stride) + 1);
                if (numThreads > 0) {
                    dim3 blockDim(blockSize);
                    dim3 gridDim((numThreads + blockSize - 1) / blockSize);

                    upSweep << <gridDim, blockDim >> > (pow2_n, dev_data, stride);
                    checkCUDAError("upSweep failed!");
                }
            }

            hipMemset(dev_data + pow2_n - 1, 0, sizeof(int));
            checkCUDAError("hipMemset last element failed!");

            for (int stride = pow2_n >> 1; stride > 0; stride >>= 1) {
                int numThreads = pow2_n >> (ilog2(stride) + 1);
                if (numThreads > 0) {
                    dim3 blockDim(blockSize);
                    dim3 gridDim((numThreads + blockSize - 1) / blockSize);

                    downSweep << <gridDim, blockDim >> > (pow2_n, dev_data, stride);
                    checkCUDAError("downS failed!");
                }
            }

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from device failed!");

            hipFree(dev_data);

            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
